#include <iostream>
#include <random>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void add(int *a, int *b, int *c, int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		c[index] = a[index] + b[index];
	}
}


int main() {

	int n;

	cout << "Enter size of vector: ";
	cin >> n;

	int *a = new int[n];
	int *b = new int[n];
	int *c = new int[n];

	cout << "Enter vector 1: " << endl;
	for (int i = 0; i < n; i++) {
		cin >> a[i];
	}

	cout << "Enter vector 2: " << endl;
	for (int i = 0; i < n; i++) {
		cin >> b[i];
	}

	int *x, *y, *z;

	hipMalloc(&x, n * sizeof(int));
	hipMalloc(&y, n * sizeof(int));
	hipMalloc(&z, n * sizeof(int));

	hipMemcpy(x, a, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(y, b, n * sizeof(int), hipMemcpyHostToDevice);
	add<<<n, 1>>>(x, y, z, n);
	hipMemcpy(c, z, n * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < n; i++) {
		cout << a[i] << " ";
	}
	cout << endl;
	for (int i = 0; i < n; i++) {
		cout << b[i] << " ";
	}
	cout << endl;
	for (int i = 0; i < n; i++) {
		cout << c[i] << " ";
	}
	cout << endl;
	delete[] a;
	delete[] b;
	delete[] c;

	hipFree(x);
	hipFree(y);
	hipFree(z);

	return 0;
}
